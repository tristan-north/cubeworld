#include "hip/hip_runtime.h"
#include "cutil_math.h"  // required for float3 vector math
#include <Windows.h>  // This needs to be included before openGL stuff
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <cuda_gl_interop.h>
#include <glm/glm.hpp>
#include <fstream>

#include "common.h"
#include "kernel.cuh"
#include "camera.h"

#define M_PI 3.14159265359f

// output buffer
float3 *g_rgbBuffer;

struct Ray {
	float3 orig;	// ray origin
	float3 dir;		// ray direction	
	__device__ Ray(float3 o_, float3 d_) : orig(o_), dir(d_) {}
};

// required to convert colour to a format that OpenGL can display  
union Colour  // 4 bytes = 4 chars = 1 float
{
	float c;
	uchar4 components;
};

// helper functions
inline __device__ float3 minf3(float3 a, float3 b){ return make_float3(a.x < b.x ? a.x : b.x, a.y < b.y ? a.y : b.y, a.z < b.z ? a.z : b.z); }
inline __device__ float3 maxf3(float3 a, float3 b){ return make_float3(a.x > b.x ? a.x : b.x, a.y > b.y ? a.y : b.y, a.z > b.z ? a.z : b.z); }
inline __device__ float minf1(float a, float b){ return a < b ? a : b; }
inline __device__ float maxf1(float a, float b){ return a > b ? a : b; }

__constant__ float aspectRatio = float(WIDTH) / HEIGHT;
__constant__ float3 gridMin = { -50.0f, 0.0f, -50.0f };
__constant__ float3 gridMax = { 50.0f, 100.0f, 50.0f };
//__constant__ uint gridRes = 5;
//__constant__ uint grid[5][5][5] = { 
//{ { 0, 0, 0, 0, 0 }, { 0, 0, 0, 0, 0 }, { 1, 0, 1, 0, 0 }, { 0, 1, 0, 0, 0 }, { 0, 0, 0, 0, 0 } },
//{ { 0, 0, 0, 0, 0 }, { 0, 0, 1, 0, 0 }, { 0, 0, 1, 0, 0 }, { 0, 0, 0, 0, 0 }, { 0, 0, 0, 0, 0 } },
//{ { 0, 0, 0, 0, 0 }, { 0, 0, 0, 0, 0 }, { 0, 0, 1, 0, 0 }, { 0, 0, 1, 0, 0 }, { 0, 1, 0, 0, 0 } },
//{ { 0, 0, 0, 0, 0 }, { 0, 1, 0, 0, 0 }, { 0, 1, 0, 0, 0 }, { 0, 0, 0, 0, 0 }, { 0, 1, 1, 0, 0 } },
//{ { 0, 0, 0, 0, 0 }, { 0, 0, 0, 1, 0 }, { 0, 0, 0, 0, 1 }, { 1, 0, 0, 0, 0 }, { 0, 0, 0, 0, 0 } },
//};
__constant__ uint gridRes = 10;
__constant__ uint grid[10][10][10] = {
	{ { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 1, 1, 0, 0, 0, 0 }, { 1, 0, 0, 0, 1, 1, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 1, 1, 0, 0, 0, 0 }, { 1, 0, 0, 0, 1, 1, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 0, 0, 0, 0, 0, 0, 0, 0, 0 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
	{ { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 }, { 1, 1, 1, 1, 1, 1, 1, 1, 1, 1 } },
};


__device__ bool sphere_intersect(const Ray &r, float &t, const glm::vec3 pos, const float radius) {
	// Ray/sphere intersection
	// Quadratic formula required to solve ax^2 + bx + c = 0 
	// Solution x = (-b +- sqrt(b*b - 4ac)) / 2a
	// Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0 

	glm::vec3 op = pos - glm::vec3(r.orig.x, r.orig.y, r.orig.z);
	float b = dot(op, glm::vec3(r.dir.x, r.dir.y, r.dir.z));
	float disc = b*b - dot(op, op) + radius*radius; // discriminant
	if (disc<0) return false; else disc = sqrtf(disc);

	t = b - disc;
	if (t > EPSILON) return true;
	t = b + disc;
	if (t > EPSILON) return true;
	return false;
}

__device__ glm::vec3 point_on_sphere(const glm::vec3 pos, float radius, float rand1, float rand2) {
	// From first example in http://mathworld.wolfram.com/SpherePointPicking.html
	float theta = 2 * M_PI * rand1;
	float phi = acos(2 * rand2 - 1);

	glm::vec3 p;
	p.x = cos(theta) * sin(phi);
	p.y = sin(theta) * sin(phi);
	p.z = cos(phi);
	
	return p * radius + pos;
}

__device__ void create_orthonormal_coords(const float3 &w, float3 &u, float3 &v) {
	if (fabs(w.x) > .1f)
		u = cross(make_float3(0.0f, 1.0f, 0.0f), w);
	else
		u = cross(make_float3(1.0f, 0.0f, 0.0f), w);
	u = normalize(u);
	v = cross(u, w);
}

__device__ bool box_intersect(const Ray &r, float &t, const float3 min, const float3 max) {

	// This division should be precomputed if it ends up getting called a lot for the same ray dir.
	// Need to make sure r.dir isn't 0 before doing the divide.
	float3 rayDirSafe;
	if (fabs(r.dir.x) < EPSILON) rayDirSafe.x = EPSILON; else rayDirSafe.x = r.dir.x;
	if (fabs(r.dir.y) < EPSILON) rayDirSafe.y = EPSILON; else rayDirSafe.y = r.dir.y;
	if (fabs(r.dir.z) < EPSILON) rayDirSafe.z = EPSILON; else rayDirSafe.z = r.dir.z;
	float3 rayDirInv = { 1.0f / rayDirSafe.x, 1.0f / rayDirSafe.y, 1.0f / rayDirSafe.z };

	float3 tmin = (min - r.orig) * rayDirInv;
	float3 tmax = (max - r.orig) * rayDirInv;

	float3 real_min = minf3(tmin, tmax);
	float3 real_max = maxf3(tmin, tmax);

	float minmax = minf1(minf1(real_max.x, real_max.y), real_max.z);
	float maxmin = maxf1(maxf1(real_min.x, real_min.y), real_min.z);

	if (minmax >= maxmin) {
		if (maxmin < EPSILON) return false;
		if (t > LARGE_VAL) t = LARGE_VAL;
		t = maxmin;
		return true;
	}
	else return false;
}

__device__ bool ground_intersect(const Ray &ray, float &t, float3 &color, float3 &normal) {
	float denom = dot(make_float3(0.0f, 1.0f, 0.0f), ray.dir);
	if (denom < EPSILON) {
		float3 p0l0 = make_float3(0.0f, 0.0f, 0.0f) - ray.orig;
		t = dot(p0l0, make_float3(0.0f, 1.0f, 0.0f)) / denom;
		if (t > LARGE_VAL) t = LARGE_VAL;
		color = { 0.9f, 0.9f, 0.9f };
		normal = { 0.0f, 1.0f, 0.0f };
		return (t >= 0);
	}
	return false;
}


__device__ inline bool grid_intersect(const Ray &inRay, float &t, float3 &color, float3 &normal) {
	Ray ray = inRay;
	if (fabs(inRay.dir.x) < EPSILON) ray.dir.x = EPSILON; else ray.dir.x = inRay.dir.x;
	if (fabs(inRay.dir.y) < EPSILON) ray.dir.y = EPSILON; else ray.dir.y = inRay.dir.y;
	if (fabs(inRay.dir.z) < EPSILON) ray.dir.z = EPSILON; else ray.dir.z = inRay.dir.z;
	//assert(fabs(ray.dir.x) > EPSILON/2); assert(fabs(ray.dir.y) > EPSILON/2); assert(fabs(ray.dir.z) > EPSILON/2);

	const float3 cellSize = { (gridMax.x - gridMin.x) / gridRes, (gridMax.y - gridMin.y) / gridRes, (gridMax.z - gridMin.z) / gridRes };

	float bboxIsecDist = 0;
	// Check if ray starts inside bbox.
	if (! ((ray.orig.x > gridMin.x && ray.orig.x < gridMax.x) && (ray.orig.y > gridMin.y && ray.orig.y < gridMax.y) && (ray.orig.z > gridMin.z && ray.orig.z < gridMax.z))) {
		// If not inside find if and where ray hits grid bbox
		if (!box_intersect(ray, bboxIsecDist, gridMin, gridMax)) {
			return false;  // If doesn't hit the bbox at all just return
		}
	}
	const float3 gridIsecPoint = ray.orig + ray.dir*bboxIsecDist;

	// rayOGridspace is the ray origin position relative to the grid origin position. Ie rayOGrid is in "grid space".
	const float3 rayOGridspace = { gridIsecPoint.x - gridMin.x, gridIsecPoint.y - gridMin.y, gridIsecPoint.z - gridMin.z };
	// This is the ray origin position in "cell space". Ie if rayOCell.x is 2.5,
	// the ray starts in the middle of the 3rd cell in x.
	const float3 rayOCellspace = { rayOGridspace.x / cellSize.x, rayOGridspace.y / cellSize.y, rayOGridspace.z / cellSize.z };

	uint3 cellIndex;
	cellIndex.x = floor(rayOCellspace.x); cellIndex.x = clamp(cellIndex.x, uint(0), gridRes - 1);
	cellIndex.y = floor(rayOCellspace.y); cellIndex.y = clamp(cellIndex.y, uint(0), gridRes - 1);
	cellIndex.z = floor(rayOCellspace.z); cellIndex.z = clamp(cellIndex.z, uint(0), gridRes - 1);
	//color = make_float3(float(cellIndex.x) / (gridRes-1), float(cellIndex.y) / (gridRes-1), float(cellIndex.z) / (gridRes-1));  return true;

	// deltaT is the distance between cell border intersections for each axis
	const float deltaTx = fabs(cellSize.x / ray.dir.x);
	const float deltaTy = fabs(cellSize.y / ray.dir.y);
	const float deltaTz = fabs(cellSize.z / ray.dir.z);

	// tx, ty and tz are how far along the ray needs to be travelled to get to the
	// next (based on current t) cell in x, next cell in y and next cell in z.
	// Whichever is smallest will be the next intersection.
	float tx = ((cellIndex.x + 1) * cellSize.x - rayOGridspace.x) / ray.dir.x;
	if (ray.dir.x < 0)
		tx = (cellIndex.x * cellSize.x - rayOGridspace.x) / ray.dir.x;
	float ty = ((cellIndex.y + 1) * cellSize.y - rayOGridspace.y) / ray.dir.y;
	if (ray.dir.y < 0)
		ty = (cellIndex.y * cellSize.y - rayOGridspace.y) / ray.dir.y;
	float tz = ((cellIndex.z + 1) * cellSize.z - rayOGridspace.z) / ray.dir.z;
	if (ray.dir.z < 0)
		tz = (cellIndex.z * cellSize.z - rayOGridspace.z) / ray.dir.z;

	// Used to either increment or decrement cell index based on if ray direction is + or -.
	int stepX = ray.dir.x > 0 ? 1 : -1;
	int stepY = ray.dir.y > 0 ? 1 : -1;
	int stepZ = ray.dir.z > 0 ? 1 : -1;

	// Traverse the grid.
	t = 0;
	bool hit = false;
	const int maxCellIndex = (int)(gridRes - 1);
	normal = { 0.0f, 0.0f, 0.0f };

	while (true) {
		// Check if grid cell contents is true
		if (grid[cellIndex.x][cellIndex.y][cellIndex.z]) {
			hit = true;
			break;
		}

		// Move variables to next cell.
		if (tx <= ty && tx <= tz) {
			// tx is smallest, so we're crossing into another cell in x.
			t = tx;   // As this is the next cell boarder intersected, update t to this
			tx += deltaTx;   // update to next intersection along x
			cellIndex.x = cellIndex.x + stepX;
			normal = { 1.0f, 0.0f, 0.0f };
			normal *= -stepX;
		}
		else if (ty <= tx && ty <= tz) {
			// ty is smallest, so we're crossing into another cell in y.
			t = ty;
			ty += deltaTy;
			cellIndex.y = cellIndex.y + stepY;
			normal = { 0.0f, 1.0f, 0.0f };
			normal *= -stepY;
		}
		else if (tz <= tx && tz <= ty) {
			// tz is smallest, so we're crossing into another cell in z.
			t = tz;
			tz += deltaTz;
			cellIndex.z = cellIndex.z + stepZ;
			normal = { 0.0f, 0.0f, 1.0f };
			normal *= -stepZ;
		}

		// Break if the next cell is outside the grid.
		if (cellIndex.x > maxCellIndex || cellIndex.y > maxCellIndex || cellIndex.z > maxCellIndex) {
			t = 0;
			break;
		}
		if (cellIndex.x < 0 || cellIndex.y < 0 || cellIndex.z < 0) {
			t = 0;
			break;
		}
	}

	if (!hit) return false;

	// If the normal is still 0 here, it means the ray hit the outside of the grid so need
	// to do something else to get the normal.
	if (normal.x == 0.0f && normal.y == 0.0f && normal.z == 0.0f) {
		if (fabs(gridMin.x - gridIsecPoint.x) < EPSILON) normal = { -1.0f, 0.0f, 0.0f };
		else if (fabs(gridMax.x - gridIsecPoint.x) < EPSILON) normal = { 1.0f, 0.0f, 0.0f };
		else if (fabs(gridMin.y - gridIsecPoint.y) < EPSILON) normal = { 0.0f, -1.0f, 0.0f };
		else if (fabs(gridMax.y - gridIsecPoint.y) < EPSILON) normal = { 0.0f, 1.0f, 0.0f };
		else if (fabs(gridMin.z - gridIsecPoint.z) < EPSILON) normal = { 0.0f, 0.0f, -1.0f };
		else normal = { 0.0f, 0.0f, 1.0f };
	}

	t += bboxIsecDist;
	if (t > LARGE_VAL) t = LARGE_VAL;
	color = { 0.9, 0.3, 0.01 };
	return hit;
}


// radiance function
// compute path bounces in scene and accumulate returned color from each path sgment
__device__ float3 radiance(const Ray &camRay, hiprandState &randstate, const Light &light){ // returns ray color
	// colour mask
	float3 mask = make_float3(1.0f, 1.0f, 1.0f);
	// accumulated colour
	float3 accucolor = make_float3(0.0f, 0.0f, 0.0f);

	Ray r = camRay;
	float3 normal, surfaceColor, shadNormal, shadColor;
	float t, ndotl;
	
	//////////////////////////////
	///////////////////////////////////////////////
	//float rand1 = hiprand_uniform(&randstate);
	//float rand2 = hiprand_uniform(&randstate);
	//float3 lightCenterDir = make_float3(light.pos.x, light.pos.y, light.pos.z) - r.orig;
	//float d2 = length(lightCenterDir); d2 = d2*d2;
	//float cosThetaMax = sqrtf(1.0f - (light.radius*light.radius) / d2);

	//float costheta = (1.f - rand1) + rand1 * cosThetaMax;
	//float sintheta = sqrtf(1.f - costheta*costheta);
	//float phi = rand2 * 2.f * M_PI;
	//glm::vec3 diskPoint = { cosf(phi) * sintheta, sinf(phi) * sintheta, costheta };

	//if (sphere_intersect(camRay, t, glm::vec3(rand1, 0, 0), 0.5f)){
	//	accucolor = accucolor + make_float3(1.0f, 0.0f, 0.0f);
	//}
	//else {
	//	accucolor = accucolor + make_float3(0.0f, 1.0f, 0.0f);
	//}
	//return accucolor;
	/////////////////////////////////////////

	for (int bounces = 0; bounces < RAYDEPTH; bounces++){  // iteration (instead of recursion in CPU code)
		t = LARGE_VAL;

		// intersect ray with scene
		bool hit = grid_intersect(r, t, surfaceColor, normal);
		if (!hit) {
			hit = ground_intersect(r, t, surfaceColor, normal);
		}

		// if camera ray, test against lights
		if (bounces == 0) {
			float lightT;
			if (sphere_intersect(r, lightT, light.pos, light.radius)) {  // If the primary ray hits the light
				if (lightT < t || !hit) {  // If the distance to the light is less than to the scene hit, or if the scene wasn't hit
					accucolor = make_float3(light.color.x, light.color.y, light.color.z);
					break;
				}
			}
		}

		// if ray misses everything add sky color and break
		if (!hit) {
			accucolor += make_float3(SKY_COLOR) * mask;
			break;
		}

		// Shoot shadow ray from shading point to light
		r.orig += r.dir*t;  // Move ray origin to hit point
		r.orig += normal * 0.001;  // Shadow bias

		// Create point on a hemisphere facing the shading point
		float rand1 = hiprand_uniform(&randstate) * 0.99999f;  // This gets used in a sqrt and if it's 1 will lead to sqrt(0);
		float rand1s = sqrtf(rand1);
		float rand2 = hiprand_uniform(&randstate) * 2 * M_PI;

		// Move point to world space
		float3 u, v, w;
		float3 vecToLightCenter = make_float3(light.pos.x, light.pos.y, light.pos.z) - r.orig;
		w = normalize(vecToLightCenter);
		create_orthonormal_coords(w, u, v);
		float3 pointOnLight = make_float3(light.pos.x, light.pos.y, light.pos.z) + normalize(u*cos(rand2)*rand1s + v*sin(rand2)*rand1s - w*sqrtf(1 - rand1)) * light.radius;
		 
		float3 vecToLightSample = pointOnLight - r.orig;
		r.dir = normalize(vecToLightSample);
		ndotl = max(dot(normal, r.dir), 0.0f);

		if (ndotl > 0)  // Don't bother tracing a shadow ray if the light is hitting the backface
			hit = grid_intersect(r, t, shadColor, shadNormal);
		else {
			hit = true; 
			t = 0.0f;
		}

		// Test if the distance to the light is closer than the distance to the hit point, in which case it's not shadowed
		if (hit) {
			if (t > length(vecToLightSample)) hit = false;
		}
		if (pointOnLight.y < 0.0f) hit = true; // If light sample is under the ground plane, consider it shadowed

		// If not shadowed
		if (!hit) {
			float3 lightColor = make_float3(light.color.x, light.color.y, light.color.z);
			//lightColor *= 1 / (length(vecToLight)*length(vecToLight));  // Distance falloff
			
			// Generate pdf. This was reference http://graphics.pixar.com/library/PhysicallyBasedLighting/paper.pdf
			float3 lightCenterDir = make_float3(light.pos.x, light.pos.y, light.pos.z) - r.orig;
			float d2 = length(lightCenterDir);
			d2 = d2*d2;
			if (d2 - light.radius*light.radius < EPSILON) // The shading point is inside the light
				break;
	
			float cosThetaMax = sqrtf(1.0f - (light.radius*light.radius) / d2);
			// Technically pdf should be 1 / 2*M_PI * (1.0f - cosThetaMax) but the 1 / 2*M_PI would just get cancelled out later
			float pdf = 1.0f - cosThetaMax;
			accucolor += (surfaceColor * ndotl * lightColor * pdf) * mask;
		}
		mask *= surfaceColor;

		// Set up indirect ray dir for next ray depth loop
		// Create new cosine weighted ray
		// create 2 random numbers
		float r1 = 2 * M_PI * hiprand_uniform(&randstate);
		float r2 = hiprand_uniform(&randstate) * 0.99999f;  // This gets used in a sqrt and if it's 1 will lead to sqrt(0)
		float r2s = sqrtf(r2);

		// compute orthonormal coordinate frame uvw with hitpoint as origin 
		w = normal;
		create_orthonormal_coords(w, u, v);
		
		// compute cosine weighted random ray direction on hemisphere 
		r.dir = normalize(u*cos(r1)*r2s + v*sin(r1)*r2s + w*sqrtf(1 - r2));
	}

	
	// add radiance up to a certain ray depth
	// return accumulated ray colour after all bounces are computed
	return accucolor;
}


__global__ void render_kernel(float3 *output, uint hashedpassnumber, float3 camOrig, float3 camDir, float3 camUp, Light light){

	// assign a CUDA thread to every pixel by using the threadIndex
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= WIDTH || y >= HEIGHT) return;

	// global threadId, see richiesams blogspot
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// create random number generator, see RichieSams blogspot
	hiprandState randState; // state of the random number generator, to prevent repetition
	hiprand_init(hashedpassnumber + threadId, 0, 0, &randState);

	float3 pixelcol = { 0.0f, 0.0f, 0.0f }; // final pixel color     
	
	for (int s = 0; s < SPP; s++){
		float3 d = camOrig + camDir * 1.5;  // Move point out from cam origin along cam dir. This distance controls the FOV.
		float rand1 = hiprand_uniform(&randState) - 0.5f;
		float rand2 = hiprand_uniform(&randState) - 0.5f;

		d += (cross(camDir, camUp) * ((x + rand1) / WIDTH - 0.5f) * aspectRatio);  // Move our ray origin along right vector an amount depending on x value of pixel
		d += (camUp* ((y + rand2) / HEIGHT - 0.5f));  // Move our ray origin along up vector an amount depending on y value of pixel
		d = normalize(d - camOrig);  // Get vector between new point and cam orig.

		pixelcol += radiance(Ray(camOrig, d), randState, light)*(1. / SPP);
	}

	// Gamma correction
	pixelcol.x = powf(pixelcol.x, 1 / 2.2);
	pixelcol.y = powf(pixelcol.y, 1 / 2.2);
	pixelcol.z = powf(pixelcol.z, 1 / 2.2);

	// This tone mapping is the one unreal engine uses. It incudes gamma correction. Could try changing the coefficients to get different looks.
	//pixelcol.x = pixelcol.x / (pixelcol.x + 0.187f) * 1.035f;
	//pixelcol.y = pixelcol.y / (pixelcol.y + 0.187f) * 1.035f;
	//pixelcol.z = pixelcol.z / (pixelcol.z + 0.187f) * 1.035f;
	
	// Convert to unsigned char for openGL.
	Colour fcolour;
	fcolour.components = make_uchar4((unsigned char)clamp(pixelcol.x * 255.0f, 0.0f, 255.0f),
									(unsigned char)clamp(pixelcol.y * 255.0f, 0.0f, 255.0f),
									(unsigned char)clamp(pixelcol.z * 255.0f, 0.0f, 255.0f), 1);
	// store pixel coordinates and pixelcolour in OpenGL readable outputbuffer
	int i = (HEIGHT - y - 1)*WIDTH + x; // pixel index
	output[i] = make_float3(x, y, fcolour.c);
}


void cudaInit(GLuint vbo) {

	//register VBO with CUDA
	cudaGLRegisterBufferObject(vbo);
}

void cudaCleanup() {
	hipFree(g_rgbBuffer);
}

void launchKernel(GLuint vbo, uint rand, Camera* cam, Light* light) {
	// map vertex buffer object for access by CUDA 
	cudaGLMapBufferObject((void**)&g_rgbBuffer, vbo);

	dim3 blockSize(32, 32, 1);
	dim3 gridSize((int)ceil((float)WIDTH / blockSize.x), (int)ceil((float)HEIGHT / blockSize.y));

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// launch CUDA path tracing kernel, pass in a hashed seed based on number of passes
	render_kernel << < gridSize, blockSize >> >(g_rgbBuffer, rand,
												make_float3(cam->position().x, cam->position().y, cam->position().z),
												make_float3(cam->forward().x, cam->forward().y, cam->forward().z),
												make_float3(cam->up().x, cam->up().y, cam->up().z),
												*light);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	//fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(hipPeekAtLastError()));
	//printf("Kernel time:  %.3f ms \n", time);

	hipDeviceSynchronize();

	// unmap buffer
	cudaGLUnmapBufferObject(vbo);
}